#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <iomanip>
#include <cstring>
#include <hip/hip_runtime.h>
#include <cstdio>

//C++ timers
#include <chrono>

#define BLOCK_SIZE 16

#define CUDA_WARN(XXX) \
    do { if (XXX != hipSuccess) std::cerr << "CUDA Error: " << \
        hipGetErrorString(XXX) << ", at line " << __LINE__ \
        << std::endl; hipDeviceSynchronize(); } while (0)

typedef std::chrono::high_resolution_clock Clock;
typedef std::chrono::nanoseconds nanoseconds;

typedef struct {
	
	//Dimensions of A
	int M;
	int N;

	//Scalars
	double alpha;
	double beta;
	
	//First dimension of A 
	int lda;

	//Increments for X and Y
	int incX;
	int incY;

	//Array A and vectors X, Y;
	double *A, *X, *Y;
} dgemv_data;


// BASIC DGEMM METHOD
void dgemm(int N,       double alpha,
		   double *X,   double *Y,
           double beta, double *A) {

 	
  	for (int i = 0; i < N; ++i) {
    	for (int j = 0; j < N; ++j) {
            double tmp = 0;
            for (int k = 0; k < N; ++k) {
                tmp += X[i * N + k] * Y[k * N + j];
            }
            A[i * N + j] = alpha * tmp + beta * A[i * N + j];
        }
    }
}

 
// CUDA DGEMM W/O SHARED MEMORY
__global__ void dgemm_cuda (int N,       double alpha,
						    double *X,   double *Y,
							double beta, double *A) {

  	int i = blockDim.y * blockIdx.y + threadIdx.y;
  	int j = blockDim.x * blockIdx.x + threadIdx.x;
  
  	if (i < N && j < N) {
  
  		double temp = 0;
  		for (int k = 0; k < N; ++k) {
    		temp += X[i * N + k] * Y[k * N + j];
  		}
    
    	A[i * N + j] = alpha * temp + beta * A[i * N + j];
	}
    
}

 
// CUDA DGEMM W/O SHARED MEMORY
__global__ void dgemm_cuda_shared (int N,       double alpha,
						    	   double *X,   double *Y,
								   double beta, double *A) {

	// Create Shared Memory Arrays
    __shared__ double Xshared[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double Yshared[BLOCK_SIZE][BLOCK_SIZE];

    //Setup i and j
    int i = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int j = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    double temp = 0;

    for (int s = 0; s < gridDim.x; ++s) {

        int index = i * N + s * BLOCK_SIZE + threadIdx.x;
        if(index >= N*N) 
            Xshared[threadIdx.y][threadIdx.x] = 0;
        else 
            Xshared[threadIdx.y][threadIdx.x] = X[index];
        

        index = (s * BLOCK_SIZE + threadIdx.y) * N + j;
        if(index >= N*N) 
            Yshared[threadIdx.y][threadIdx.x] = 0;
        else 
            Yshared[threadIdx.y][threadIdx.x] = Y[index];
        
        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k) 
            temp += Xshared[threadIdx.y][k] * Yshared[k][threadIdx.x];
        
        __syncthreads();
    }
    
    if(i < N && j < N) {
        A[i * N + j] = temp * alpha + beta * A[i * N + j];
    }

}


double* createMatrix(int M, int N) {
	double *A = new double[M*N];

	int i; 
	#pragma omp parallel for private(i)
	for(i = 0; i < M*N; i++) {
		A[i] = ((double) rand() / (RAND_MAX)); //Random number from 0 to 1
	}
		
	
	return A;
}

 double* createVector(int len) {
	double *A = new double[len];
	
	int i;
	#pragma omp parallel for private(i)
	for(i = 0; i < len; i++) {
		A[i] = ((double) rand() / (RAND_MAX));
	}
		
	
	return A;
}

dgemv_data* generateRandomData(int M, int N) {

	dgemv_data *data = new dgemv_data();
	data->M = M;
	data->N = N;
	data->A = new double[M*N];
	data->alpha = (double) (rand() % 10)  + ((double) rand() / (RAND_MAX)); 
	data->beta = (double) (rand() % 10)  + ((double) rand() / (RAND_MAX)); 
	data->lda = M; //Not used for dgemm
	data->X = createMatrix(N,N);
	data->Y = createMatrix(N,N);
	data->incX = 1; //Not used for dgemm
	data->incY = 1; //not used for dgemm

	return data;
}

dgemv_data* copyData(dgemv_data* data) {
	dgemv_data* copy = new dgemv_data();

	copy->M = data->N;
	copy->N = data->N;	
	copy->alpha = data->alpha;
	copy->beta = data->beta;
	copy->lda = data->lda;
	copy->incX = data->incX;
	copy->incY = data->incY;
	copy->X = new double[data->N * data->N];
	copy->Y = new double[data->N * data->N];
	copy->A = new double[data->N * data->N];
	memcpy (copy->X, data->X, data->N * data->N * sizeof(double));
	memcpy (copy->Y, data->Y, data->N * data->N * sizeof(double));
	memcpy (copy->A, data->A, data->N * data->N * sizeof(double));

	return copy;
}

bool compareMatrices(double *A,  double *B, int length) {
	int i;
	for(i = 0; i < length; i++) {
		//To account for doubleing pt error, check if greater than some epsilon
		if(abs(A[i] - B[i]) > 0.000001)  { 
			std::cout <<"i: " << i << " A[i]: " << A[i] << " B[i]: " << B[i] << std::endl;
			return false;
		}
	}

	return true;
}

void freeDataStruct(dgemv_data* data) {
	delete[](data->A);
	delete[](data->X);
	delete[](data->Y);
}

void testOutput(dgemv_data *data, dgemv_data* test_data) {
		dgemm(test_data->N,   test_data->alpha,
		       test_data->X,   test_data->Y,
		       test_data->beta,test_data->A);
		
		if(compareMatrices(data->A, test_data->A, data->N * data->N)) {
			std::cout << "Output: PASSED" << std::endl;
		} else {
			std::cout << "Output: FAILED" << std::endl;
		}
}

void printTimeTaken(unsigned long ns) {
	std::cout << std::fixed;
    std::cout << std::setprecision(10)
			  << "Time taken: " 
			  << ns 
			  << " ns or " 
			  << (double) ns/1000000000.0
			  << " s\n"
			  << std::endl;
}

int main(int argc, char **argv) {

	//Clock for C++
	if (argc != 3) {
		std::cout << "Invalid set of arguments.\n"
				  << "Usage: ./dgemv [Testing Off/On(0/1)] [size N]"
				  << std::endl;
		exit(-1);
	}

	//Get user arguments
	int M,N;
	bool test;

	//Square matrix
	if(argc == 3) {
		M = atoi(argv[2]);
		N = M;
	} 
	
	test = (atoi(argv[1]) > 0) ? true : false;


	//Feed random seed
	srand(time(NULL));

	//Generate the data
	dgemv_data *unModifiedData = generateRandomData(M,N); //DO not run functions on this
	


	/************************************************************************/
	//Run DGEMVT Serial Version
	
	//Get data sets to run with and test with (Since the arrays are modified)
	dgemv_data *data = copyData(unModifiedData);

	dgemv_data *serialTestData = copyData(unModifiedData);
	std::cout << "-----------------------------------------------" << std::endl;
	std::cout << "Running Serial Version of DGEMVT" << std::endl;
	auto start = Clock::now();
	dgemm(data->N, data->alpha, data->X,
		   data->Y, data->beta,  data->A);

	auto end = Clock::now();
	unsigned long ns = (unsigned long) std::chrono::duration_cast<nanoseconds>(end-start).count();
	printTimeTaken(ns);

	//Test the output
	if(test) {
		testOutput(data, serialTestData);
	}


	//Congifure CUDA blocksize and dim grid/block
    size_t gridR = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    size_t gridC = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
	dim3 dimGrid(gridC,gridR);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);		



	/************************************************************************/
	//Run DGEMVT CUDA Version
	
	
	//Get data sets to run with and test with (Since the arrays are modified)
	dgemv_data *cuData = copyData(unModifiedData);
	dgemv_data *cuTestData = copyData(unModifiedData);
	
	//Double arrays for the GPU
	double *cuA, *cuX, *cuY;

	//Allocate space and copy data into GPU allocated arrays
	CUDA_WARN(hipMalloc(&cuA, N * N * sizeof(double)));
	CUDA_WARN(hipMalloc(&cuX, N * N * sizeof(double)));	
	CUDA_WARN(hipMalloc(&cuY, N * N * sizeof(double)));	
	
	CUDA_WARN(hipMemcpy(cuA, cuData->A, N*N*sizeof(double), hipMemcpyHostToDevice));	
	CUDA_WARN(hipMemcpy(cuX, cuData->X, N*N*sizeof(double), hipMemcpyHostToDevice));	
	CUDA_WARN(hipMemcpy(cuY, cuData->Y, N*N*sizeof(double), hipMemcpyHostToDevice));	

	std::cout << "-----------------------------------------------" << std::endl;
	std::cout << "Running CUDA Version of DGEMVT" << std::endl;
	start = Clock::now();
	dgemm_cuda<<<dimGrid, dimBlock>>>
			   (N,cuData->alpha, cuX,
                cuY, cuData->beta, cuA);
	CUDA_WARN(hipDeviceSynchronize());
	end = Clock::now();
	ns = (unsigned long) std::chrono::duration_cast<nanoseconds>(end-start).count();

	CUDA_WARN(hipGetLastError());

	
	//Copy the cude result back
	hipMemcpy(cuData->A, cuA, N*N*sizeof(double), hipMemcpyDeviceToHost);

	//Print the time taken
	printTimeTaken(ns);

	//Test the output
	if(test) {
		testOutput(cuData, cuTestData);
	}

	/************************************************************************/
	//Run DGEMVT CUDA Shared Memory Version	
	
	//Get data sets to run with and test with (Since the arrays are modified)
	dgemv_data *cuDataShared = copyData(unModifiedData);
	dgemv_data *cuTestDataShared = copyData(unModifiedData);
	
	//Double arrays for the GPU
	double *scuA, *scuX, *scuY;

	//Allocate space and copy data into GPU allocated arrays
	CUDA_WARN(hipMalloc(&scuA, N * N * sizeof(double)));
	CUDA_WARN(hipMalloc(&scuX, N * N * sizeof(double)));	
	CUDA_WARN(hipMalloc(&scuY, N * N * sizeof(double)));	
	
	CUDA_WARN(hipMemcpy(scuA, cuDataShared->A, N*N*sizeof(double), hipMemcpyHostToDevice));	
	CUDA_WARN(hipMemcpy(scuX, cuDataShared->X, N*N*sizeof(double), hipMemcpyHostToDevice));	
	CUDA_WARN(hipMemcpy(scuY, cuDataShared->Y, N*N*sizeof(double), hipMemcpyHostToDevice));	

	std::cout << "-----------------------------------------------" << std::endl;
	std::cout << "Running CUDA Shared Memory Version of DGEMVT" << std::endl;
	start = Clock::now();
	dgemm_cuda_shared<<<dimGrid, dimBlock>>>
			   (N,cuDataShared->alpha, scuX,
                scuY, cuDataShared->beta, scuA);
	CUDA_WARN(hipDeviceSynchronize());
	end = Clock::now();
	ns = (unsigned long) std::chrono::duration_cast<nanoseconds>(end-start).count();

	CUDA_WARN(hipGetLastError());

	
	//Copy the cude result back
	hipMemcpy(cuDataShared->A, scuA, N*N*sizeof(double), hipMemcpyDeviceToHost);

	//Print the time taken
	printTimeTaken(ns);

	//Test the output
	if(test) {
		testOutput(cuDataShared, cuTestDataShared);
	}


	return 0;
}
