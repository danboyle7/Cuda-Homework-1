#include <iostream>
#include <cstdlib>
#include <iomanip>
#include <cstring>
#include <hip/hip_runtime.h>
#include <cstdio>

#define CUDA_WARN(XXX) \
      do { if (XXX != hipSuccess) std::cerr << "CUDA Error: " << \
          hipGetErrorString(XXX) << ", at line " << __LINE__ \
          << std::endl; hipDeviceSynchronize(); } while (0)


int main() {
	double *h1 = new double[5];
	double *h2 = new double[5]; 
    double *d;

	h1[0] = 7;
	h1[1] = 4;	
	h1[2] = 1;
	h1[3] = 8;
	h1[4] = 9;
	
	std::cout << "1) Mallocing Space and copying from host to device" << std::endl;	
	CUDA_WARN(hipMalloc(&d, 5*sizeof(double)));
	CUDA_WARN(hipMemcpy(d, h1, 5*sizeof(double), hipMemcpyHostToDevice));

	std::cout << "\n2) Copying back from device to host"  << std::endl;
	CUDA_WARN(hipMemcpy(h2, d, 5*sizeof(double), hipMemcpyDeviceToHost));

	std::cout << "\n3) Copied data (back on host): " << std::endl;
	for(int i = 0; i < 5; i++) {
		std::cout << h2[i] << std::endl;
	} 
 
	
	return 0;
}

